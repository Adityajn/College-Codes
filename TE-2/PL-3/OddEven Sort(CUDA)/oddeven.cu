
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define SIZE 10
__global__ void sort(int* A){
	
	bool even=true;
	__shared__ bool swappedOdd;
	__shared__ bool swappedEven;
	__syncthreads();

	swappedEven=true;
	swappedOdd=true;
	do{
		if(even){
			swappedEven=false;
			__syncthreads();
			if(threadIdx.x<blockDim.x/2){
				if(A[threadIdx.x*2]>A[threadIdx.x*2+1]){
					int temp=A[threadIdx.x*2];
					A[threadIdx.x*2]=A[threadIdx.x*2+1];
					A[threadIdx.x*2+1]=temp;
					swappedEven=true;	
				}	
			}
		}
		else{
			swappedOdd=false;
			__syncthreads();
			if(threadIdx.x<blockDim.x/2){
				if(A[threadIdx.x*2+1]>A[threadIdx.x*2+2]){
					int temp=A[threadIdx.x*2+2];
					A[threadIdx.x*2+2]=A[threadIdx.x*2+1];
					A[threadIdx.x*2+1]=temp;
					swappedOdd=true;	
				}	
			}
		}
		__syncthreads();
		even=!even;
		__syncthreads();
	}
	while(swappedEven || swappedOdd);
}

int main(){
	int *A,*B,*A_d;

	//initialize local mem
	A=(int*)malloc(SIZE*sizeof(int));
	B=(int*)malloc(SIZE*sizeof(int));
	printf("Enter unsorted items :");
	for(int i=0;i<SIZE;i++){
		scanf("%d",&A[i]);
	}

	//initialize device mem
	hipMalloc(&A_d,SIZE*sizeof(int));
	hipMemcpy(A_d,A,SIZE*sizeof(int),hipMemcpyHostToDevice);

	sort <<< 1,SIZE >>>(A_d);

	hipMemcpy(B,A_d,SIZE*sizeof(int),hipMemcpyDeviceToHost);
	
	printf("\nSorted Array is :");
	for(int i=0;i<SIZE;i++){
		printf("%d   ",B[i]);
	}

}