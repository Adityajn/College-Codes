
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define SIZE 16

__global__ void multiply(float *A, float *B, float* C, int rc){
	
	int col=gridDim.x*blockIdx.x + threadIdx.x; 
	int row=gridDim.y*blockIdx.y + threadIdx.y;

	float sum=0;
	for(int i=0;i<rc;i++){
		sum+=A[(row*rc)+i]*B[(rc*i)+col];
		if(row==0 && col==0){
			printf("%f %f %d %d\n",A[(row*rc)+i],B[(rc*i)+col],(row*rc)+i,(rc*i)+col);
		}
	}
	
	C[(row*rc)+col]=sum;
}


int main(){
	float *a,*b,*c;
	float *a_d,*b_d,*c_d;
	int rc=(int)sqrt(SIZE);
	//initialize local mem
	a=(float*)malloc(SIZE*sizeof(float));
	b=(float*)malloc(SIZE*sizeof(float));
	c=(float*)malloc(SIZE*sizeof(float));
	printf("Enter First Matrix(%d x %d) :\n",rc,rc);
	for(int i=0;i<rc;i++){
		for(int j=0;j<rc;j++){
			scanf("%f",&a[i*rc+j]);
		}
		printf("\n");
	}
	printf("Enter Second Matrix(%d x %d) :\n",rc,rc);
	for(int i=0;i<rc;i++){
		for(int j=0;j<rc;j++){
			scanf("%f",&b[i*rc+j]);
		}
		printf("\n");
	}

	//initialize cuda memory
	hipMalloc(&a_d,SIZE*sizeof(float));	
	hipMalloc(&b_d,SIZE*sizeof(float));
	hipMalloc(&c_d,SIZE*sizeof(float));

	hipMemcpy(a_d,a,SIZE*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(b_d,b,SIZE*sizeof(float),hipMemcpyHostToDevice);

	dim3 gridSize(2,2);
	dim3 blockSize(2,2);

	multiply <<<gridSize,blockSize>>> (a_d,b_d,c_d,rc);

	hipMemcpy(c,c_d,SIZE*sizeof(float),hipMemcpyDeviceToHost);

	printf("Product Matrix(%d x %d) :\n",rc,rc);
	for(int i=0;i<rc;i++){
		for(int j=0;j<rc;j++){
			printf("%f\t",c[i*rc+j]);
		}
		printf("\n");
	}


	free(a);free(b);free(c);
	hipFree(a_d);hipFree(b_d);hipFree(c_d);
	return 0;
}