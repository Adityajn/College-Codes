#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#define SIZE 20
using namespace std;

__global__ void compute(int* a,int* b, int* c){

	__shared__ int  temp[SIZE];
	int index=blockIdx.x*SIZE+threadIdx.x;	
	temp[index]=a[index]*b[index];

	__syncthreads();

	for(int i =0;i<SIZE/2;i++){
		c[i]=temp[i]+temp[i+SIZE/2];
	}

}


int main(){
	//declare variables
	int *A,*B,*C;
	int n= SIZE * sizeof(int);
	int *A_d,*B_d,*C_d;

	//initialize local variables
	A=(int*)malloc(n);
	B=(int*)malloc(n);
	C=(int*)malloc(n/2);
	for(int i=0;i<2*SIZE;i++){
		A[i]=rand()%10;
		B[i]=rand()%10;
	}
	for(int i=0;i<10;i++){
		cout<<A[i]<<"  "<<A[10+i]<<"      "<<B[i]<<"  "<<B[i+10]<<endl;
	}

	//initialize GPU memory
	hipMalloc(&A_d,n);				//hipMalloc(**ptr,SIZE);
	hipMalloc(&B_d,n);
	hipMalloc(&C_d,n/2);
	hipMemcpy(A_d,A,SIZE*sizeof(int),hipMemcpyHostToDevice);					//hipMemcpy( *dst,*source, size, type);
	hipMemcpy(B_d,B,SIZE*sizeof(int),hipMemcpyHostToDevice);

	compute<<<1,20>>>(A_d,B_d,C_d);

	hipMemcpy(C,C_d,SIZE*sizeof(int)/2,hipMemcpyDeviceToHost);

	for(int i=0;i<10;i++){
			cout<<"     "<<C[i]<<sendl;
	}	

	free(A);free(B);free(C);
//	hipFree(A_d);hipFree(B_d);hipFree(C_d);
	return 0;
}