
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define BLOCKS 3
#define THREADS 10
#define SIZE 30


__global__ void compute(int* a,int* b, int* c){
	__shared__ int temp[SIZE];
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	temp[x] = a[x] * b[x];
	c[threadIdx.x]=0;
	__syncthreads();
	atomicAdd(&c[threadIdx.x],temp[x]);
}

int main(){
	
	int i;
	//declare local variables
	int *a,*b,*c;
	//declare device variables
	int *a_d,*b_d,*c_d;
	clock_t stime,ftime;

	//initialize local variables
	a = (int*) malloc(SIZE*sizeof(int));
	b = (int*) malloc(SIZE*sizeof(int));
	c = (int*) malloc(THREADS*sizeof(int));
	for(i=0;i<SIZE;i++){
		a[i]=rand()%10;
		b[i]=rand()%10;
	}
	//print inputs arrays
	printf("\tA\t\t\tB\t\n");
	for(i=0;i<THREADS;i++){
		printf("%di + %dj + %dk \t\t\t%di + %dj +%dk\n",a[i],a[i+10],a[i+20],b[i],b[i+10],b[i+20]);
	}

	//initialize device variables
	hipMalloc(&a_d,SIZE*sizeof(int));
	hipMalloc(&b_d,SIZE*sizeof(int));
	hipMalloc(&c_d,THREADS*sizeof(int));
	hipMemcpy(a_d,a,SIZE*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(b_d,b,SIZE*sizeof(int),hipMemcpyHostToDevice);
	
	//start kernel
	stime=clock();
	compute<<<BLOCKS,THREADS>>>(a_d,b_d,c_d);
	ftime=clock();
	//get results
	hipMemcpy(c,c_d,THREADS*sizeof(int),hipMemcpyDeviceToHost);

	printf("\t\tC\n");
	for(i=0;i<THREADS;i++){
		printf("\t\t%d\n",c[i]);
	}
	printf("Execution Time %e :",((double)ftime-stime));
	return 0;
}